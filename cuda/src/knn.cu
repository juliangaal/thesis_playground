#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

class Point
{
public:
    Point() = default;

    __host__ __device__
    Point(double x, double y) : x(x), y(y)
    {};
    double x, y;
};

__device__ Point d_query_point;

template<typename T>
struct device_sort
{
    typedef T first_argument_type;
    typedef T second_argument_type;
    typedef bool result_type;

    __host__ __device__ bool operator()(const T &a, const T &b) const
    {
        // Not concerned with actual distances, so skip the sqrt
        double norm_a =
            (a.x - d_query_point.x) * (a.x - d_query_point.x) + (a.y - d_query_point.y) * (a.y - d_query_point.y);
        double norm_b =
            (b.x - d_query_point.x) * (b.x - d_query_point.x) + (b.y - d_query_point.y) * (b.y - d_query_point.y);
        return norm_a < norm_b;
        //return true;
    }
};

// This simple "hello world" example implements the kNearestNeighbors example on a set of example 2D points.
int main(void)
{
    thrust::device_vector<Point> d_points;
    thrust::host_vector<Point> h_points;

    Point query_point = Point(0, 0);

    h_points.push_back(Point(2, 0));
    h_points.push_back(Point(1, 0));
    h_points.push_back(Point(0, 10));
    h_points.push_back(Point(5, 5));
    h_points.push_back(Point(2, 5));

    hipMemcpyToSymbol(HIP_SYMBOL(d_query_point), &query_point, sizeof(Point));
    // transfer to device
    d_points = h_points;

    thrust::sort(d_points.begin(), d_points.end(), device_sort<Point>());

    // transfer results to host
    h_points = d_points;

    for (const auto p: h_points)
    {
        cout << p.x << ", " << p.y << endl;
    }

    return 0;
}
