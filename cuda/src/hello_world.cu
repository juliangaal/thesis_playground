
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(void)
{
	printf("Hello from GPU\n");
}

int main(void)
{
	printf("Hello World from CPU!\n");
	
	hello_from_gpu<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}
