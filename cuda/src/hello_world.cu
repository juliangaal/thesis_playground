#include "hip/hip_runtime.h"
#include "hello_world.cuh"
#include <stdio.h>

__global__ void hello_from_gpu_impl()
{
    printf("Hello from GPU\n");
}

void cuda::hello_from_gpu()
{
    hello_from_gpu_impl<<<1, 10>>>();
}


