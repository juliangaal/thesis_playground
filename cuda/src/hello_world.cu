#include "hip/hip_runtime.h"
#include "hello_world.h"
#include <stdio.h>

__global__ void hello_from_gpu_krnl()
{
    printf("Hello from GPU\n");
}

void cuda::hello_from_gpu()
{
    hello_from_gpu_krnl<<<1, 10>>>();
}
