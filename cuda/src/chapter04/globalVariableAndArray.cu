#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <algorithm>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

#define SIZE 100

__device__ float devData;
__device__ unsigned int size;
__device__ float data[SIZE];

__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    // alter the value
    devData += 2.0f;
}

__global__ void chapter4exercise1()
{
    auto t = threadIdx.x;
    printf("%d %d\n", blockIdx.x, t);
    if (t < size)
    {
        data[t] = 100;
    }
}

void checkResult(float *hostRef, float *gpuRef, const int size, float value)
{
    double epsilon = 1.0E-8;

    for (int i = 0; i < size; i++)
    {
        assert(abs(hostRef[i] - value) < epsilon);
    }

    printf("Arrays match.\n\n");
}


int main(void)
{
    // initialize the global variable
    float value = 3.14f;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host:   copied %f to the global variable devData\n", value);

    auto n = SIZE;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(size), &n, sizeof(unsigned int)));
    printf("Host:   copied %d to the global variable size\n", n);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // copy the global variable back to the host
    CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:   the value changed by the kernel  to %f\n", value);

    float h_A[SIZE] = { 0 };
    std::fill(h_A, h_A+SIZE, value);
    for (int i = 0; i < n; i++)
    {
        assert(h_A[i] == value);
    }

    // copy data from host to device
    hipMemcpyToSymbol(HIP_SYMBOL(data), h_A, n * sizeof(float));
    dim3 block(SIZE);
    dim3 grid(1);
    chapter4exercise1<<<grid, block>>>();
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(data), n * sizeof(float)));

    checkResult(h_A, data, n, 100);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
