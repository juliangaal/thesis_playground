#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <iostream>

#include "common.h"

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            return;
        }
    }

    printf("Arrays match.\n\n");
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx,
                     const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];

        }

        ia += nx;
        ib += nx;
        ic += nx;
    }
}

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF ) / 10.0f;
    }
}


__global__ void sumMatrixOnGPU1D(const float *A, const float *B, float *C, int nx, int ny)
{
    // data point in kernel refers to this block coordinate in 2D matrix
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y;
    auto idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // setup size
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // initialize arrays (host)
    float *h_A, *h_B, *hostRef, *gpuRef;
    auto start = seconds();
    h_A     = (float*)malloc(nBytes);
    h_B     = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef  = (float*)malloc(nBytes);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    auto end = seconds() - start;
    printf("host memory alloc took %f sec\n", end);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    // run host code
    start = seconds();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    end = seconds() - start;
    printf("sumMatrixOnHost elapsed %f sec\n", end);

    // initialize arrays (device) and malloc device global memory
    float *d_A, *d_B, *d_C;
    start = seconds();
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy host data to device data
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
    end = seconds() - start;
    printf("Cuda memory alloc: %f sec\n", end);

    // launch kernel and wait for results
    dim3 block(128);
    dim3 grid((nx + block.x - 1) / block.x, ny);

    start = seconds();
    sumMatrixOnGPU1D<<<grid, block>>>(d_A, d_B, d_C, nx, ny);
    CHECK(hipDeviceSynchronize());
    end = seconds() - start;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, end);

    // copy device data to host
    CHECK(hipMemcpy(gpuRef, d_C, nBytes,hipMemcpyDeviceToHost));
    end = seconds() - start;
    printf("Result in host memory after %f sec", end);

    // compare
    checkResult(hostRef, gpuRef, nxy);

    // free memory device
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free memory host
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}

